#include "hip/hip_runtime.h"
#include "RainbowRayTracer.cuh"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <cmath>
#include <chrono>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define WIDTH 1024    // width of the generated image
#define HEIGHT 1024   // height of the generated image
#define CHANNEL_NUM 4 // the number of color channels per pixel
#define MAX_BOUNCES 4 // the maximum number of times a light ray can bounce or interact with the sphere during the ray tracing process

/**
 * @brief Computes perfect reflection direction (R = I - 2·N(N·I))
 *
 * Implements the standard reflection equation for ideal mirrors:
 * R = I - 2·N(N·I) where:
 * - I: Incident direction (unit vector)
 * - N: Surface normal (unit vector)
 * - R: Reflected direction (unit vector)
 *
 * @param I Incident direction (must be normalized)
 * @param N Surface normal (must be normalized)
 * @return Reflected direction (normalized)
 *
 * Note: Equivalent to GLSL reflect() function.
 *       Maintains input vector normalization.
 */
__device__ Vec3 reflect(const Vec3 &I, const Vec3 &N)
{
    return I - N * 2.0f * I.dot(N);
}

/**
 * @brief Computes refraction direction using Snell's law.
 *
 * Implements:
 * n₁·sinθ₁ = n₂·sinθ₂
 * Returns zero vector for total internal reflection (when sinθ₂ > 1)
 *
 * @param incident Incident direction (unit vector)
 * @param normal   Surface normal (unit vector)
 * @param n1       Refractive index of origin medium (η₁)
 * @param n2       Refractive index of destination medium (η₂)
 * @return Refracted direction (unit vector if valid) or
 *         zero vector for total internal reflection
 */
__device__ Vec3 refract(const Vec3 &incident, const Vec3 &normal, float n1, float n2)
{
    float ratio = n1 / n2;                                         // Refractive index ratio
    float cosI = -fmaxf(-1.0f, fminf(1.0f, incident.dot(normal))); // Clamp the dot product to [-1, 1]
    float sinT2 = ratio * ratio * (1.0f - cosI * cosI);            // Sine squared of the refraction angle

    if (sinT2 > 1.0f)
        return Vec3(0, 0, 0); // Total internal reflection: return zero vector

    float cosT = sqrtf(1.0f - sinT2);                         // Cosine of the refraction angle
    return incident * ratio + normal * (ratio * cosI - cosT); // Refraction direction
}

/**
 * @brief Fresnel equations for reflection and refraction blending
 *
 * Implements the Fresnel equations for unpolarized light:
 * R = ½(R_∥ + R_⊥) where:
 * R_∥ = ((n₂·cosθ₁ - n₁·cosθ₂)/(n₂·cosθ₁ + n₁·cosθ₂))²
 * R_⊥ = ((n₁·cosθ₁ - n₂·cosθ₂)/(n₁·cosθ₁ + n₂·cosθ₂))²
 *
 * @param incident Incident ray direction (unit vector)
 * @param normal   Surface normal (unit vector)
 * @param n1       Refractive index of origin medium
 * @param n2       Refractive index of destination medium
 * @return Reflectance ∈ [0.0,1.0] where:
 *         0.0 = full transmission
 *         1.0 = total internal reflection (when sinθ₂ > 1.0)
 */
__device__ float fresnel(const Vec3 &incident, const Vec3 &normal, float n1, float n2)
{
    float cosI = -fmaxf(-1.0f, fminf(1.0f, incident.dot(normal))); // Cosine of the angle of incidence
    float sinT2 = (n1 / n2) * (n1 / n2) * (1.0f - cosI * cosI);    // Sine squared of the angle of refraction

    if (sinT2 > 1.0f)
        return 1.0f; // Total internal reflection: return maximum reflectance

    float cosT = sqrtf(1.0f - sinT2);                                                 // Cosine of the angle of refraction
    float rParallel = ((n2 * cosI) - (n1 * cosT)) / ((n2 * cosI) + (n1 * cosT));      // Reflection for parallel polarization
    float rPerpendicular = ((n1 * cosI) - (n2 * cosT)) / ((n1 * cosI) + (n2 * cosT)); // Reflection for perpendicular polarization

    return (rParallel * rParallel + rPerpendicular * rPerpendicular) / 2.0f; // Average reflection
}

/**
 * @brief Tests ray-sphere intersection using geometric solution.
 *
 * Solves quadratic equation: t² + 2b·t + c = 0
 * where:
 *   b = (origin - center) · direction
 *   c = (origin - center)² - radius²
 *
 * @param origin  Ray starting point (world space)
 * @param dir     Normalized ray direction
 * @param sphere  Sphere (center + radius) to test
 * @param t       Output: Distance to nearest intersection if found
 * @return true if ray hits sphere (with t > 0), false otherwise
 */
__device__ bool intersectRaySphere(const Vec3 &origin, const Vec3 &dir, const Sphere &sphere, float &t)
{
    Vec3 oc = origin - sphere.center;                     // Vector from ray origin to sphere center
    float b = 2.0f * oc.dot(dir);                         // Dot product term for the quadratic equation
    float c = oc.dot(oc) - sphere.radius * sphere.radius; // Constant term for the quadratic equation
    float discriminant = b * b - 4.0f * c;                // Discriminant of the quadratic equation

    // If the discriminant is positive, there are real intersections
    if (discriminant > 0)
    {
        t = (-b - sqrtf(discriminant)) / 2.0f; // Calculate the intersection distance
        return t > 0;                          // If t is positive, the intersection occurs in the direction of the ray
    }

    // No intersection
    return false;
}

/**
 * @brief Computes wavelength-dependent refractive index for water/air interface.
 *
 * Uses an empirical approximation of the Sellmeier dispersion formula:
 * n(λ) = 1.31477 + 0.0108148/log₁₀(0.00690246λ)
 *
 * @param wavelength Light wavelength in nanometers [380,780]
 * @return Refractive index (n) with:
 *         - n ≈ 1.33 for visible spectrum
 *         - Higher dispersion at shorter wavelengths (blue/violet)
 */
__device__ float wavelengthToRefraction(float wavelength)
{
    return 1.31477f + 0.0108148f / log10f(0.00690246f * wavelength);
}

/**
 * @brief Converts a visible light wavelength to sRGB color with gamma correction.
 *
 * Implements a piecewise linear approximation of the visible spectrum:
 * 380-440nm: Violet to Blue
 * 440-490nm: Blue to Cyan
 * 490-510nm: Cyan to Green
 * 510-580nm: Green to Yellow
 * 580-645nm: Yellow to Red
 * 645-780nm: Red
 *
 * @param wavelength Input wavelength in nanometers [380,780]
 * @return RGBA color with:
 *         - Gamma correction (γ=0.8)
 *         - Intensity falloff at spectrum edges
 *         - Alpha always 255 (opaque)
 *
 * Note: Returns black for wavelengths outside visible range.
 */
__device__ Color wavelengthToRGB(float wavelength)
{
    float gamma = 0.8;
    float intensityMax = 255;
    float factor;
    float r = 0.0, g = 0.0, b = 0.0;

    // Color Mapping
    if (wavelength >= 380 && wavelength < 440)
    {
        // violet to blue
        r = -(wavelength - 440) / (440 - 380);
        b = 1.0;
    }
    else if (wavelength >= 440 && wavelength < 490)
    {
        // blue to cyan
        g = (wavelength - 440) / (490 - 440);
        b = 1.0;
    }
    else if (wavelength >= 490 && wavelength < 510)
    {
        // cyan to green
        g = 1.0;
        b = -(wavelength - 510) / (510 - 490);
    }
    else if (wavelength >= 510 && wavelength < 580)
    {
        // green to yellow
        r = (wavelength - 510) / (580 - 510);
        g = 1.0;
    }
    else if (wavelength >= 580 && wavelength < 645)
    {
        // yellow to orange
        r = 1.0;
        g = -(wavelength - 645) / (645 - 580);
    }
    else if (wavelength >= 645 && wavelength < 781)
    {
        // orange to red
        r = 1.0;
    }

    // Intensity Factor
    if (wavelength >= 380 && wavelength < 420)
        // violet
        factor = 0.3 + 0.7 * (wavelength - 380) / (420 - 380);
    else if (wavelength >= 420 && wavelength < 701)
        // blue to red
        factor = 1.0;
    else if (wavelength >= 701 && wavelength < 781)
        // red
        factor = 0.3 + 0.7 * (780 - wavelength) / (780 - 700);
    else
        // no color - wavelength is outside the visible spectrum
        factor = 0.0;

    return {
        static_cast<unsigned char>(intensityMax * pow(r * factor, gamma)),
        static_cast<unsigned char>(intensityMax * pow(g * factor, gamma)),
        static_cast<unsigned char>(intensityMax * pow(b * factor, gamma)),
        255};
}

/**
 * @brief Traces a light ray through a sphere, simulating wavelength-dependent refraction and reflection.
 *
 * Implements:
 * - Snell's Law for refraction
 * - Fresnel equations for reflectance
 * - Chromatic dispersion via wavelength-to-refraction mapping
 * - Up to maxBounces interactions
 *
 * @param origin      Ray starting position (world space)
 * @param dir         Normalized ray direction
 * @param sphere      Sphere to intersect with (center + radius)
 * @param maxBounces  Maximum ray interactions (refractions+reflections)
 * @param wavelength  Light wavelength in nm (380-780) for dispersion effects
 *
 * @return Accumulated color from all ray interactions
 *
 * Note: Uses Russian roulette termination for reflectance/transmittance
 */
__device__ Color traceRay(const Vec3 &origin, const Vec3 &dir, const Sphere &sphere, int maxBounces, float wavelength, hiprandState *localRandState)
{
    Color color = {0, 0, 0, 255};
    Vec3 currentDir = dir, currentPos = origin;
    float refractiveIndex = wavelengthToRefraction(wavelength);
    bool isInside = false; // Start outside the sphere

    for (int bounce = 0; bounce < maxBounces; ++bounce)
    {
        float t;
        if (intersectRaySphere(currentPos, currentDir, sphere, t))
        {
            Vec3 hit = currentPos + currentDir * t;
            Vec3 normal = (hit - sphere.center).normalize();

            // Flip normal if ray is inside the sphere
            if (isInside)
                normal = -normal;

            // Compute Fresnel reflectance
            float n1 = isInside ? refractiveIndex : 1.0f;
            float n2 = isInside ? 1.0f : refractiveIndex;
            float reflectance = fresnel(currentDir, normal, n1, n2);

            // Compute reflection/refraction directions
            Vec3 reflected = reflect(currentDir, normal);
            Vec3 refracted = refract(currentDir, normal, n1, n2);

            // Blend color
            Color waveColor = wavelengthToRGB(wavelength);

            color.r = fminf(255.0f, color.r * (1 - reflectance) + waveColor.r * reflectance);
            color.g = fminf(255.0f, color.g * (1 - reflectance) + waveColor.g * reflectance);
            color.b = fminf(255.0f, color.b * (1 - reflectance) + waveColor.b * reflectance);

            // Update ray position/direction
            currentPos = hit;

            // Decide reflection or refraction
            if (refracted.dot(refracted) > 0)
            {
                float randVal = hiprand_uniform(localRandState); // Get a random float in [0, 1)
                currentDir = (randVal < reflectance) ? reflected : refracted;
                isInside = !isInside;
            }
            else
            {
                currentDir = reflected;
            }
        }
        else
            break; // Ray escaped
    }

    return color;
}

/**
 * @brief CUDA kernel to render a scene pixel by pixel.
 *
 * This kernel performs ray tracing to compute the color of each pixel in an image.
 * It simulates light propagation and interaction with a sphere in the scene. The kernel
 * computes the wavelength of light for each pixel and traces rays from a light source,
 * interacting with the sphere. The color is determined based on reflection, refraction,
 * and the Fresnel effect.
 *
 * @param pixels Pointer to the output pixel buffer. Each pixel is represented by 4 channels (RGBA).
 * @param sphere Pointer to a Sphere object in the scene. This sphere is the object being rendered.
 * @param light Pointer to a Light object in the scene. This light source is used to compute the light direction.
 * @param wavelengthStart The starting wavelength (in nm) for the rainbow effect.
 * @param wavelengthEnd The ending wavelength (in nm) for the rainbow effect.
 */
__global__ void renderKernel(unsigned char *pixels, Sphere *sphere, Light *light, float wavelengthStart, float wavelengthEnd)
{
    // Initialize the random number generator state for this thread
    hiprandState state;

    // Loop over every pixel in the image (height and width of the render)
    for (int y = 0; y < HEIGHT; ++y)
    {
        for (int x = 0; x < WIDTH; ++x)
        {
            // Calculate the index in the pixel buffer for the current pixel
            int index = (y * WIDTH + x) * CHANNEL_NUM;

            // Compute the wavelength for the current pixel based on its y-coordinate (for a rainbow effect)
            float wavelength = wavelengthEnd - (wavelengthEnd - wavelengthStart) * (y / float(HEIGHT));

            // Calculate the direction of light from the light source to the sphere
            Vec3 lightDir = (sphere->center - light->pos).normalize();

            // Get the color of the pixel corresponding to the current wavelength
            Color baseColor = wavelengthToRGB(wavelength);

            // Trace the ray from the light source and compute the resulting color at the hit point
            Color color = traceRay(light->pos, lightDir, *sphere, MAX_BOUNCES, light->wavelength, &state);

            // Blend the calculated color with the base color (for a rainbow effect)
            color.r = fminf(255, color.r + baseColor.r);
            color.g = fminf(255, color.g + baseColor.g);
            color.b = fminf(255, color.b + baseColor.b);

            // Store the final color in the output pixel buffer (RGBA)
            pixels[index + 0] = color.r; // Red channel
            pixels[index + 1] = color.g; // Green channel
            pixels[index + 2] = color.b; // Blue channel
            pixels[index + 3] = 255;     // Alpha channel (full opacity)
        }
    }
}

int main()
{
    std::cout << "Starting CUDA SEQUENTIAL ray tracing..." << std::endl;
    auto start = std::chrono::high_resolution_clock::now();

    unsigned char *d_pixels, *pixels = new unsigned char[WIDTH * HEIGHT * CHANNEL_NUM];
    hipMalloc(&d_pixels, WIDTH * HEIGHT * CHANNEL_NUM);

    Sphere h_sphere = {{WIDTH / 2.0f, HEIGHT / 2.0f, 50.0f}, 40.0f};
    Light h_light = {{WIDTH / 2.0f, HEIGHT, 100.0f}, {0, -1, -0.5f}};

    Sphere *d_sphere;
    Light *d_light;
    hipMalloc(&d_sphere, sizeof(Sphere));
    hipMalloc(&d_light, sizeof(Light));
    hipMemcpy(d_sphere, &h_sphere, sizeof(Sphere), hipMemcpyHostToDevice);
    hipMemcpy(d_light, &h_light, sizeof(Light), hipMemcpyHostToDevice);

    renderKernel<<<1, 1>>>(d_pixels, d_sphere, d_light, 380.0f, 750.0f);

    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, WIDTH * HEIGHT * CHANNEL_NUM, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();

    stbi_write_png("rainbow_cuda_seq.png", WIDTH, HEIGHT, CHANNEL_NUM, pixels, WIDTH * CHANNEL_NUM);

    std::cout << "CUDA ray tracing completed." << std::endl;
    std::cout << "Image dimensions: " << WIDTH << "x" << HEIGHT << std::endl;
    std::cout << "Max bounces: " << MAX_BOUNCES << std::endl;
    std::cout << "Execution time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms.\n";

    hipFree(d_pixels);
    hipFree(d_sphere);
    hipFree(d_light);
    delete[] pixels;

    std::cout << "Output saved to rainbow_cuda_seq.png\n";
    std::cout << "Press Enter to exit...";
    std::cin.ignore();
    return 0;
}
