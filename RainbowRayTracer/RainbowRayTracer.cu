#include "hip/hip_runtime.h"
#include "RainbowRayTracer.cuh"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <cmath>
#include <chrono>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define WIDTH 1024    // width of the generated image
#define HEIGHT 1024   // height of the generated image
#define CHANNEL_NUM 4 // the number of color channels per pixel
#define MAX_BOUNCES 4 // the maximum number of times a light ray can bounce or interact with the sphere during the ray tracing process

/**
 * @brief Computes perfect reflection direction (R = I - 2·N(N·I))
 *
 * Implements the standard reflection equation for ideal mirrors:
 * R = I - 2·N(N·I) where:
 * - I: Incident direction (unit vector)
 * - N: Surface normal (unit vector)
 * - R: Reflected direction (unit vector)
 *
 * @param I Incident direction (must be normalized)
 * @param N Surface normal (must be normalized)
 * @return Reflected direction (normalized)
 *
 * Note: Equivalent to GLSL reflect() function.
 *       Maintains input vector normalization.
 */
__device__ Vec3 reflect(const Vec3 &I, const Vec3 &N)
{
    return I - N * 2.0f * I.dot(N);
}

/**
 * @brief Computes refraction direction using Snell's law.
 *
 * Implements:
 * n₁·sinθ₁ = n₂·sinθ₂
 * Returns zero vector for total internal reflection (when sinθ₂ > 1)
 *
 * @param incident Incident direction (unit vector)
 * @param normal   Surface normal (unit vector)
 * @param n1       Refractive index of origin medium (η₁)
 * @param n2       Refractive index of destination medium (η₂)
 * @return Refracted direction (unit vector if valid) or
 *         zero vector for total internal reflection
 */
__device__ Vec3 refract(const Vec3 &incident, const Vec3 &normal, float n1, float n2)
{
    float ratio = n1 / n2;                                         // Refractive index ratio
    float cosI = -fmaxf(-1.0f, fminf(1.0f, incident.dot(normal))); // Clamp the dot product to [-1, 1]
    float sinT2 = ratio * ratio * (1.0f - cosI * cosI);            // Sine squared of the refraction angle

    if (sinT2 > 1.0f)
        return Vec3(0, 0, 0); // Total internal reflection: return zero vector

    float cosT = sqrtf(1.0f - sinT2);                         // Cosine of the refraction angle
    return incident * ratio + normal * (ratio * cosI - cosT); // Refraction direction
}

/**
 * @brief Fresnel equations for reflection and refraction blending
 *
 * Implements the Fresnel equations for unpolarized light:
 * R = ½(R_∥ + R_⊥) where:
 * R_∥ = ((n₂·cosθ₁ - n₁·cosθ₂)/(n₂·cosθ₁ + n₁·cosθ₂))²
 * R_⊥ = ((n₁·cosθ₁ - n₂·cosθ₂)/(n₁·cosθ₁ + n₂·cosθ₂))²
 *
 * @param incident Incident ray direction (unit vector)
 * @param normal   Surface normal (unit vector)
 * @param n1       Refractive index of origin medium
 * @param n2       Refractive index of destination medium
 * @return Reflectance ∈ [0.0,1.0] where:
 *         0.0 = full transmission
 *         1.0 = total internal reflection (when sinθ₂ > 1.0)
 */
__device__ float fresnel(const Vec3 &incident, const Vec3 &normal, float n1, float n2)
{
    float cosI = -fmaxf(-1.0f, fminf(1.0f, incident.dot(normal))); // Cosine of the angle of incidence
    float sinT2 = (n1 / n2) * (n1 / n2) * (1.0f - cosI * cosI);    // Sine squared of the angle of refraction

    if (sinT2 > 1.0f)
        return 1.0f; // Total internal reflection: return maximum reflectance

    float cosT = sqrtf(1.0f - sinT2);                                                 // Cosine of the angle of refraction
    float rParallel = ((n2 * cosI) - (n1 * cosT)) / ((n2 * cosI) + (n1 * cosT));      // Reflection for parallel polarization
    float rPerpendicular = ((n1 * cosI) - (n2 * cosT)) / ((n1 * cosI) + (n2 * cosT)); // Reflection for perpendicular polarization

    return (rParallel * rParallel + rPerpendicular * rPerpendicular) / 2.0f; // Average reflection
}

/**
 * @brief CUDA kernel to render a scene pixel by pixel.
 *
 * This kernel performs ray tracing to compute the color of each pixel in an image.
 * It simulates light propagation and interaction with a sphere in the scene. The kernel
 * computes the wavelength of light for each pixel and traces rays from a light source,
 * interacting with the sphere. The color is determined based on reflection, refraction,
 * and the Fresnel effect.
 *
 * @param pixels Pointer to the output pixel buffer. Each pixel is represented by 4 channels (RGBA).
 * @param sphere Pointer to a Sphere object in the scene. This sphere is the object being rendered.
 * @param light Pointer to a Light object in the scene. This light source is used to compute the light direction.
 * @param wavelengthStart The starting wavelength (in nm) for the rainbow effect.
 * @param wavelengthEnd The ending wavelength (in nm) for the rainbow effect.
 */
__global__ void renderKernel(unsigned char *pixels, Sphere *sphere, Light *light, float wavelengthStart, float wavelengthEnd)
{
    // Initialize the random number generator state for this thread
    hiprandState state;

    // Loop over every pixel in the image (height and width of the render)
    for (int y = 0; y < HEIGHT; ++y)
    {
        for (int x = 0; x < WIDTH; ++x)
        {
            // Calculate the index in the pixel buffer for the current pixel
            int index = (y * WIDTH + x) * CHANNEL_NUM;

            // Compute the wavelength for the current pixel based on its y-coordinate (for a rainbow effect)
            float wavelength = wavelengthEnd - (wavelengthEnd - wavelengthStart) * (y / float(HEIGHT));

            // Calculate the direction of light from the light source to the sphere
            Vec3 lightDir = (sphere->center - light->pos).normalize();

            // Get the color of the pixel corresponding to the current wavelength
            Color baseColor = wavelengthToRGB(wavelength);

            // Trace the ray from the light source and compute the resulting color at the hit point
            Color color = traceRay(light->pos, lightDir, *sphere, MAX_BOUNCES, light->wavelength, &state);

            // Blend the calculated color with the base color (for a rainbow effect)
            color.r = fminf(255, color.r + baseColor.r);
            color.g = fminf(255, color.g + baseColor.g);
            color.b = fminf(255, color.b + baseColor.b);

            // Store the final color in the output pixel buffer (RGBA)
            pixels[index + 0] = color.r; // Red channel
            pixels[index + 1] = color.g; // Green channel
            pixels[index + 2] = color.b; // Blue channel
            pixels[index + 3] = 255;     // Alpha channel (full opacity)
        }
    }
}

int main()
{
    std::cout << "Starting CUDA SEQUENTIAL ray tracing..." << std::endl;
    auto start = std::chrono::high_resolution_clock::now();

    unsigned char *d_pixels, *pixels = new unsigned char[WIDTH * HEIGHT * CHANNEL_NUM];
    hipMalloc(&d_pixels, WIDTH * HEIGHT * CHANNEL_NUM);

    Sphere h_sphere = {{WIDTH / 2.0f, HEIGHT / 2.0f, 50.0f}, 40.0f};
    Light h_light = {{WIDTH / 2.0f, HEIGHT, 100.0f}, {0, -1, -0.5f}};

    Sphere *d_sphere;
    Light *d_light;
    hipMalloc(&d_sphere, sizeof(Sphere));
    hipMalloc(&d_light, sizeof(Light));
    hipMemcpy(d_sphere, &h_sphere, sizeof(Sphere), hipMemcpyHostToDevice);
    hipMemcpy(d_light, &h_light, sizeof(Light), hipMemcpyHostToDevice);

    renderKernel<<<1, 1>>>(d_pixels, d_sphere, d_light, 380.0f, 750.0f);

    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, WIDTH * HEIGHT * CHANNEL_NUM, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();

    stbi_write_png("rainbow_cuda_seq.png", WIDTH, HEIGHT, CHANNEL_NUM, pixels, WIDTH * CHANNEL_NUM);

    std::cout << "CUDA ray tracing completed." << std::endl;
    std::cout << "Image dimensions: " << WIDTH << "x" << HEIGHT << std::endl;
    std::cout << "Max bounces: " << MAX_BOUNCES << std::endl;
    std::cout << "Execution time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms.\n";

    hipFree(d_pixels);
    hipFree(d_sphere);
    hipFree(d_light);
    delete[] pixels;

    std::cout << "Output saved to rainbow_cuda_seq.png\n";
    std::cout << "Press Enter to exit...";
    std::cin.ignore();
    return 0;
}
